/******************************************************************************
MIT License

Copyright (c) 2016 Antti-Pekka Hynninen
Copyright (c) 2016 Oak Ridge National Laboratory (UT-Batelle)

Permission is hereby granted, free of charge, to any person obtaining a copy
of this software and associated documentation files (the "Software"), to deal
in the Software without restriction, including without limitation the rights
to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
copies of the Software, and to permit persons to whom the Software is
furnished to do so, subject to the following conditions:

The above copyright notice and this permission notice shall be included in all
copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
SOFTWARE.
*******************************************************************************/

#include <stdio.h>
#ifdef ENABLE_NVTOOLS
#include <nvToolsExtCuda.h>
#endif
#include "CudaUtils.h"

//----------------------------------------------------------------------------------------

void set_device_array_async_T(void *data, int value, const size_t ndata, hipStream_t stream, const size_t sizeofT) {
  cudaCheck(hipMemsetAsync(data, value, sizeofT*ndata, stream));
}

void set_device_array_T(void *data, int value, const size_t ndata, const size_t sizeofT) {
  cudaCheck(hipMemset(data, value, sizeofT*ndata));
}

//----------------------------------------------------------------------------------------
//
// Jittor malloc & free
//
void cutt_malloc(void** p, size_t len, size_t& allocation) {
  cudaCheck(hipMalloc(p, len));
}

void cutt_free(void* p, size_t len, size_t& allocation) {
  cudaCheck(hipFree(p));
}

void (*custom_cuda_malloc)(void** p, size_t len, size_t& allocation) = NULL;

void (*custom_cuda_free)(void* p, size_t len, size_t& allocation) = NULL;

//----------------------------------------------------------------------------------------
//
// Allocate gpu memory
// pp = memory pointer
// len = length of the array
//
void allocate_device_T(void **pp, const size_t len, const size_t sizeofT) {
  cudaCheck(hipMalloc(pp, sizeofT*len));
}

//----------------------------------------------------------------------------------------
//
// Deallocate gpu memory
// pp = memory pointer
//
void deallocate_device_T(void **pp) {
  if (*pp != NULL) {
    cudaCheck(hipFree((void *)(*pp)));
    *pp = NULL;
  }

}

//----------------------------------------------------------------------------------------
//
// Jittor allocate gpu memory
// pp = memory pointer
// len = length of the array
//
void jit_allocate_device_T(void **pp, const size_t len, const size_t sizeofT, size_t& allocation) {
  if (custom_cuda_malloc==NULL){
    cutt_malloc(pp, sizeofT*len, allocation);
  }else custom_cuda_malloc(pp, sizeofT*len, allocation);
}

//----------------------------------------------------------------------------------------
//
// Jittor deallocate gpu memory
// pp = memory pointer
//
void jit_deallocate_device_T(void **pp, const size_t len, const size_t sizeofT, size_t& allocation) {
  if (*pp != NULL) {
    if (custom_cuda_free==NULL){
      cutt_free((void *)(*pp), sizeofT*len, allocation);
    }else custom_cuda_free((void *)(*pp), sizeofT*len, allocation);
    *pp = NULL;
  }

}

//----------------------------------------------------------------------------------------
//
// Copies memory Host -> Device
//
void copy_HtoD_async_T(const void *h_array, void *d_array, size_t array_len, hipStream_t stream,
           const size_t sizeofT) {
  cudaCheck(hipMemcpyAsync(d_array, h_array, sizeofT*array_len, hipMemcpyHostToDevice, stream));
}

void copy_HtoD_T(const void *h_array, void *d_array, size_t array_len,
     const size_t sizeofT) {
  cudaCheck(hipMemcpy(d_array, h_array, sizeofT*array_len, hipMemcpyHostToDevice));
}

//----------------------------------------------------------------------------------------
//
// Copies memory Device -> Host
//
void copy_DtoH_async_T(const void *d_array, void *h_array, const size_t array_len, hipStream_t stream,
           const size_t sizeofT) {
  cudaCheck(hipMemcpyAsync(h_array, d_array, sizeofT*array_len, hipMemcpyDeviceToHost, stream));
}

void copy_DtoH_T(const void *d_array, void *h_array, const size_t array_len, const size_t sizeofT) {
  cudaCheck(hipMemcpy(h_array, d_array, sizeofT*array_len, hipMemcpyDeviceToHost));
}

//----------------------------------------------------------------------------------------
#ifdef ENABLE_NVTOOLS
void gpuRangeStart(const char *range_name) {
  static int color_id=0;
  nvtxEventAttributes_t att;
  att.version = NVTX_VERSION;
  att.size = NVTX_EVENT_ATTRIB_STRUCT_SIZE;
  att.colorType = NVTX_COLOR_ARGB;
  if (color_id == 0) {
    att.color = 0xFFFF0000;
  } else if (color_id == 1) {
    att.color = 0xFF00FF00;
  } else if (color_id == 2) {
    att.color = 0xFF0000FF;
  } else if (color_id == 3) {
    att.color = 0xFFFF00FF;
  }
  color_id++;
  if (color_id > 3) color_id = 0;
  att.messageType = NVTX_MESSAGE_TYPE_ASCII;
  att.message.ascii = range_name;
  nvtxRangePushEx(&att);
}

void gpuRangeStop() {
  nvtxRangePop();
}
#endif
